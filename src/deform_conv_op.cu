#include "hip/hip_runtime.h"
#ifndef TENSORFLOW_KERNELS_CONV_OPS_im2col_gpu_H_
#define TENSORFLOW_KERNELS_CONV_OPS_im2col_gpu_H_

// #if GOOGLE_CUDA

#define EIGEN_USE_GPU

#include "deform_conv.h"
#include "hip/hip_runtime.h"
#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"
#include "tensorflow/core/framework/register_types.h"
#include "tensorflow/core/framework/tensor_types.h"
#include "tensorflow/core/framework/tensor.h"
#include "tensorflow/core/platform/logging.h"

#include <algorithm>
#include <cstring>
#include <vector>
#include <stdio.h>


namespace tensorflow {

typedef Eigen::GpuDevice GPUDevice;
typedef std::vector<int32> TShape;

template <typename DType>
__device__ DType deform_im2col_bilinear(const DType* thread_input_ptr, 
										const int input_width,
        								const int thread_input_h_left,	// How many grids left in height
        								const int thread_input_w_left, 	// How many grids left in width
        								DType kernel_height_loc, 	// height w.r.t kernel start height
        								DType kernel_width_loc) {	// width w.r.t kernel start width

    int height_low = floor(kernel_height_loc);
    int width_low = floor(kernel_width_loc);
    int height_high;
    int width_high;
    if (height_low >= thread_input_h_left - 1) {
        height_high = height_low = thread_input_h_left - 1;
        kernel_height_loc = (DType)height_low;
    }
    else {
        height_high = height_low + 1;
    }

    if (width_low >= thread_input_w_left - 1) {
        width_high = width_low = thread_input_w_left - 1;
        kernel_width_loc = (DType)width_low;
    }
    else {
        width_high = width_low + 1;
    }

    DType height_low_dist = kernel_height_loc - height_low;
    DType width_low_dist = kernel_width_loc - width_low;
    DType height_high_dist = 1 - height_low_dist;
    DType width_high_dist = 1 - width_low_dist;
    //	---------
    //	| 1 | 2 |
    //	---------
    //	| 3 | 4 |
    //	---------
    DType v1 = thread_input_ptr[height_low * input_width + width_low];
    DType v2 = thread_input_ptr[height_low * input_width + width_high];
    DType v3 = thread_input_ptr[height_high * input_width + width_low];
    DType v4 = thread_input_ptr[height_high * input_width + width_high];

    DType w1 = height_high_dist * width_high_dist;
    DType w2 = height_high_dist * width_low_dist;
    DType w3 = height_low_dist * width_high_dist;
    DType w4 = height_low_dist * width_low_dist;

    DType val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
    return val;
}

// template <typename DType>
// __device__ DType deform_im2col_bilinear(const DType* bottom_data, const int data_width,
//         const int height, const int width, DType h, DType w) {

//     int h_low = floor(h);
//     int w_low = floor(w);
//     int h_high;
//     int w_high;
//     if (h_low >= height - 1) {
//         h_high = h_low = height - 1;
//         h = (DType)h_low;
//     }
//     else {
//         h_high = h_low + 1;
//     }

//     if (w_low >= width - 1) {
//         w_high = w_low = width - 1;
//         w = (DType)w_low;
//     }
//     else {
//         w_high = w_low + 1;
//     }

//     DType lh = h - h_low;
//     DType lw = w - w_low;
//     DType hh = 1 - lh, hw = 1 - lw;

//     DType v1 = bottom_data[h_low * data_width + w_low];
//     DType v2 = bottom_data[h_low * data_width + w_high];
//     DType v3 = bottom_data[h_high * data_width + w_low];
//     DType v4 = bottom_data[h_high * data_width + w_high];
//     DType w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

//     DType val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
//     return val;
// }


template <typename DType>
__global__ void deform_im2col_2d_gpu_kernel(
				const int num_kernels_per_filter, 
				const DType* batch_input_ptr, 
				const DType* batch_offset_ptr,
				const int input_height, 
				const int input_width, 
				const int kernel_height, 
				const int kernel_width,
				const int pad_height, 
				const int pad_width,
				const int stride_height, 
				const int stride_width,
				const int dilation_height,
				const int dilation_width,
				const int channel_per_deform_group,
				const int output_height, 
				const int output_width,
				DType* col_buf_3d_flatten_ptr) {

	CUDA_1D_KERNEL_LOOP(index, num_kernels_per_filter) {
		// index index of output matrix
		const int thread_output_w = index % output_width;
		const int thread_output_h = (index / output_width) % output_height;
		const int thread_channel = (index / output_width) / output_height;
		const int thread_filter_init_loc = thread_channel * kernel_height * kernel_width;

		// compute deformable group index
		const int deform_group_idx = thread_channel / channel_per_deform_group;
		const int thread_input_h = thread_output_h * stride_height - pad_height;
		const int thread_input_w = thread_output_w * stride_width - pad_width;
		// printf("%d, %d\n", pad_height, pad_width);
		const int thread_input_h_left = input_height - thread_input_h;
		const int thread_input_w_left = input_width - thread_input_w;

		DType* current_data_col_ptr = col_buf_3d_flatten_ptr + 
									  thread_filter_init_loc * output_height * output_width + 
									  thread_output_h * output_width + 
									  thread_output_w;
		const DType* thread_input_ptr = batch_input_ptr + 
										thread_channel * input_height * input_width + 
										thread_input_h * input_width + 
										thread_input_w;		
		const DType* thread_offset_ptr = batch_offset_ptr + 
										 2 * deform_group_idx * 
										 kernel_height * kernel_width *
										 output_height * output_width;
		// offset -> [2 * deform_group * kernel_width * kernel_height, output_height, output_width]

		for (int i = 0; i < kernel_height; i++) {
			for (int j = 0; j < kernel_width; j++) {
				const int offset_h_ptr = 2 * (i * kernel_width + j) * output_height * output_width + 
										 thread_output_h * output_width + 
										 thread_output_w;
				const int offset_w_ptr = 2 * (i * kernel_width + j) * output_height * output_width + 
										 output_height * output_width + 
										 thread_output_h * output_width + 
										 thread_output_w;
				const DType offset_h = thread_offset_ptr[offset_h_ptr];
				const DType offset_w = thread_offset_ptr[offset_w_ptr];
				// The datatype of following variables need to be changed into <Dtype> if the deformable conv is activated.
				const DType current_input_h = thread_input_h + i * dilation_height + offset_h;
				const DType current_input_w = thread_input_w + j * dilation_width + offset_w;
				DType val = static_cast<DType>(0);
				if (current_input_h >= 0 && current_input_w >= 0 && 
						current_input_h < input_height && current_input_w < input_width) {
					const DType kernel_height_loc = i * dilation_height + offset_h;
					const DType kernel_width_loc = j * dilation_width + offset_w;
					val = deform_im2col_bilinear(thread_input_ptr, 
												 input_width,
												 thread_input_h_left,
												 thread_input_w_left,
												 kernel_height_loc,
												 kernel_width_loc);
				}
				*current_data_col_ptr = val;
				current_data_col_ptr += output_height * output_width;
			}
		}
	}
}

namespace functor {

	inline int ProdShape(const TShape & shape, int start);
	template <typename DType>
	struct deform_im2col_2d<GPUDevice, DType> {
		void operator()(const GPUDevice& d, 				// 0 -> device
						const DType* batch_input_ptr, 		// 1 -> input data start pointer, ranging according to n
						const DType* batch_offset_ptr,
						const TShape& input_shape, 			// 2 -> input shape = [N, C, H, W]
						const TShape& col_buf_shape, 		// 3 -> shape = [filter_3d_flatten, output_rows, output_cols]
						const TShape& kernel_2d_shape,		// 4 -> kernel 2D shape
						const TShape& pad_2d_shape, 		// 5 -> padding 2D shape
						const TShape& stride_2d_shape, 		// 6 -> stride 2D shape
						const TShape& dilation_2d_shape,
						const int deform_group,
						DType* col_buf_3d_flatten_ptr) {	// 7 -> flatten col_buf_3d, shape = [1 * filter_3d_flatten_dim_ * output_2d_flatten_dim_]
			// num_axes should be smaller than block size
			int num_spatial_axes = kernel_2d_shape.size();
			int channel_per_deform_group = input_shape[1] / deform_group;
			int num_kernels_per_filter = input_shape[1] * ProdShape(col_buf_shape, 1);
			CudaLaunchConfig config = GetCudaLaunchConfig(num_kernels_per_filter, d);
			CHECK_LT(num_spatial_axes, config.thread_per_block);
			switch (num_spatial_axes) {
				case 2:
					deform_im2col_2d_gpu_kernel<DType> 
					<<<config.block_count, config.thread_per_block, 0, d.stream()>>>(
						num_kernels_per_filter, 
						batch_input_ptr, 
						batch_offset_ptr,
						input_shape[2], 
						input_shape[3], 
						kernel_2d_shape[0], 
						kernel_2d_shape[1],
						pad_2d_shape[0], 
						pad_2d_shape[1], 
						stride_2d_shape[0], 
						stride_2d_shape[1],
						dilation_2d_shape[0],
						dilation_2d_shape[1],
						channel_per_deform_group,  
						col_buf_shape[1], 
						col_buf_shape[2], 
						col_buf_3d_flatten_ptr);
					break;
				default:
					LOG(FATAL) << "im2col_nd_gpu does not support computation with "
							   << num_spatial_axes << " spatial axes";
			}
		}
	};

	inline int ProdShape(const TShape &shape, int start) {
		int64 res = 1;
		for(int i=start; i<shape.size(); i++) {
			res*=shape[i];
		}
		return res;
	}
}


#define DECLARE_GPU_SPEC(DType)                                  \
	template struct functor::deform_im2col_2d<GPUDevice, DType>; 
	
// extern template struct Copy<GPUDevice, T>;
TF_CALL_float(DECLARE_GPU_SPEC);
TF_CALL_double(DECLARE_GPU_SPEC);
// TF_CALL_half(DECLARE_GPU_SPEC);

// TF_CALL_GPU_NUMBER_TYPES(DECLARE_GPU_SPEC);
#undef DECLARE_GPU_SPEC


}

#endif  // TENSORFLOW_KERNELS_CONV_OPS_im2col_gpu_H_
